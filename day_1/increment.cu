#include <hip/hip_runtime.h>
#include <iostream>

__global__ void incrementKernel(int *data, int size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] += 2;
    }
}

int main(){
    const long int size = 5000000;
    int *hostArray = new int[size];
    int *deviceArray;

    for (int i = 0; i < size; i++) hostArray[i] = i;

    for (int i = 0; i < 30; i++) std::cout << hostArray[i] << '\t';

    std::cout << '\n';

    hipMalloc((void **)&deviceArray, size * sizeof(int));

    hipMemcpy(deviceArray, hostArray, size * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (size * threadsPerBlock - 1) / threadsPerBlock;

    incrementKernel<<<blocksPerGrid, threadsPerBlock>>>(deviceArray, size);

    hipMemcpy(hostArray, deviceArray, size * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < 30; i++) std::cout << hostArray[i] << '\t';
    
    hipFree(deviceArray);
    delete [] hostArray;

    return 0;
}