#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "../helpers/cuda_helpers.h"

#define TILE_WIDTH 2

// P = M * N
__global__ void tiledMatrixMulKernel(float* d_M, float* d_N, float* d_P, int Width){
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    float Pvalue = 0;
    for (int ph = 0; ph < Width / TILE_WIDTH; ++ph){
        Mds[ty][tx] = d_M[Row * Width + ph * TILE_WIDTH + tx];
        Nds[ty][tx] = d_N[ph * TILE_WIDTH + ph * Width + Col];
    }

}