#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vecAddKernel(float* A, float* B, float* C, int n){
    int i = blockDim.x * blockIdx.x + threadIdx.x
    if (i < n) C[i] = A[i] + B[i];
}


void vecAdd(float* h_A, float* h_B, float* h_C, int n)
{
    int size = n * sizeof(float);
    float *d_A, *d_B, *d_C;

    hipError_t err = hipMalloc((void**)&d_A, size);
    if (err != hipSuccess){
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_B, size);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_C, size);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);



}

int main(){
    const int n = 1024;
    float *h_A;
    float *h_B;
    float *h_C;

    float *d_A;
    float *d_B;
    float *d_C;
    int size = n * sizeof(float);

    

    

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}